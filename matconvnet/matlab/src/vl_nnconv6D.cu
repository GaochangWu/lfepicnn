// @file nnconv.cu
// @brief Convolution block MEX wrapper
// @author Andrea Vedaldi
// @author Max Jaderberg

/*
Copyright (C) 2014 Andrea Vedaldi and Max Jaderberg
Copyright (C) 2015 Andrea Vedaldi.

All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "bits/mexutils.h"
#include "bits/datamex.hpp"
#include "bits/nnconv6D.hpp"

#if ENABLE_GPU
#include "bits/datacu.hpp"
#endif

#include <memory>
#include <assert.h>
#include <math.h>
#include <iostream>

/* option codes */
enum {
  opt_stride = 0,
  opt_an_stride,
  opt_pad,
  opt_an_pad,
  opt_verbose,
  opt_no_der_data,
  opt_no_der_filters,
  opt_no_der_biases,
  opt_cudnn,
  opt_no_cudnn,
  opt_cudnn_workspace_limit,
  opt_transpose
} ;

/* options */
VLMXOption  options [] = {
  {"Stride",                1,   opt_stride                },
  {"StrideAngular",         1,   opt_an_stride             },
  {"Pad",                   1,   opt_pad                   },
  {"PadAngular",            1,   opt_an_pad                },
  {"Verbose",               0,   opt_verbose               },
  {"NoDerData",             0,   opt_no_der_data           },
  {"NoDerFilters",          0,   opt_no_der_filters        },
  {"NoderBiases",           0,   opt_no_der_biases         },
  {"Cudnn",                 0,   opt_cudnn                 },
  {"NoCudnn",               0,   opt_no_cudnn              },
  {"CudnnWorkSpaceLimit",   1,   opt_cudnn_workspace_limit },
  {0,                       0,   0                         }
} ;

/* ---------------------------------------------------------------- */
/*                                                          Context */
/* ---------------------------------------------------------------- */

vl::MexContext context ;

/*
 Resetting the context here resolves a crash when MATLAB quits and
 the ~Context function is implicitly called on unloading the MEX file.
 */
void atExit()
{
  context.clear() ;
}

/* ---------------------------------------------------------------- */
/*                                                       MEX driver */
/* ---------------------------------------------------------------- */

enum {
  IN_DATA = 0, IN_FILTERS, IN_BIASES, IN_DEROUTPUT, IN_END
} ;

enum {
  OUT_RESULT = 0, OUT_DERFILTERS, OUT_DERBIASES, OUT_END
} ;

void mexFunction(int nout, mxArray *out[],
                 int nin, mxArray const *in[])
{
  int strideX = 1 ;
  int strideY = 1 ;
  int strideAnX = 1 ;
  int strideAnY = 1 ;
  int padLeft = 0 ;
  int padRight = 0 ;
  int padTop = 0 ;
  int padBottom = 0 ;
  int padAnLeft = 0 ;
  int padAnRight = 0 ;
  int padAnTop = 0 ;
  int padAnBottom = 0 ;
  int numFilterGroups = 1 ;

  bool backMode = false ;
  bool hasFilters = false ;
  bool hasBiases = false ;
  bool computeDerData = true ;
  bool computeDerFilters = true ;
  bool computederBiases = true ;

  int verbosity = 0 ;
  int opt ;
  int next = IN_END ;
  mxArray const *optarg ;

  /* -------------------------------------------------------------- */
  /*                                            Check the arguments */
  /* -------------------------------------------------------------- */

  mexAtExit(atExit) ;

  if (nin < 3) {
    vlmxError(VLMXE_IllegalArgument, "There are less than three arguments.") ;
  }

  if (nin > 3 && vlmxIsString(in[3],-1)) {
    next = 3 ;
    backMode = 0 ;
  } else {
    backMode = (nin >= 4) ;
  }

  while ((opt = vlmxNextOption (in, nin, options, &next, &optarg)) >= 0) {
    switch (opt) {
      case opt_verbose :
        ++ verbosity ;
        break ;

      case opt_stride :
        if (!vlmxIsPlainMatrix(optarg,-1,-1)) {
          vlmxError(VLMXE_IllegalArgument, "STRIDE is not a plain matrix.") ;
        }
        switch (mxGetNumberOfElements(optarg)) {
          case 1:
            strideY = (int)mxGetPr(optarg)[0] ;
            strideX = strideY ; 
            break ;
          case 2:
            strideY = (int)mxGetPr(optarg)[0] ;
            strideX = (int)mxGetPr(optarg)[1] ;
            break ;
          default:
            vlmxError(VLMXE_IllegalArgument, "STRIDE has neither one nor two elements.") ;
        }
        break ;

      case opt_an_stride :
        if (!vlmxIsPlainMatrix(optarg,-1,-1)) {
          vlmxError(VLMXE_IllegalArgument, "STRIDEANGULAR is not a plain matrix.") ;
        }
        switch (mxGetNumberOfElements(optarg)) {
          case 1:
            strideAnY = (int)mxGetPr(optarg)[0] ;
            strideAnX = strideAnY ; 
            break ;
          case 2:
            strideAnY = (int)mxGetPr(optarg)[0] ;
            strideAnX = (int)mxGetPr(optarg)[1] ;
            break ;
          default:
            vlmxError(VLMXE_IllegalArgument, "STRIDEANGULAR has neither one nor two elements.") ;
        }
        break ;

      case opt_pad :
        if (!vlmxIsPlainMatrix(optarg,-1,-1)) {
          vlmxError(VLMXE_IllegalArgument, "PAD is not a plain matrix.") ;
        }
        switch (mxGetNumberOfElements(optarg)) {
          case 1:
            padLeft = (int)mxGetPr(optarg)[0] ;
            padRight = padLeft ;
            padTop = padLeft ;
            padBottom = padLeft ;
            break ;
          case 4:
            padTop = (int)mxGetPr(optarg)[0] ;
            padBottom = (int)mxGetPr(optarg)[1] ;
            padLeft = (int)mxGetPr(optarg)[2] ;
            padRight = (int)mxGetPr(optarg)[3] ;
            break ;
          default:
            vlmxError(VLMXE_IllegalArgument, "PAD has neither one nor four elements.") ;
        }
        break ;

      case opt_an_pad :
        if (!vlmxIsPlainMatrix(optarg,-1,-1)) {
          vlmxError(VLMXE_IllegalArgument, "PADANGULAR is not a plain matrix.") ;
        }
        switch (mxGetNumberOfElements(optarg)) {
          case 1:
            padAnLeft = (int)mxGetPr(optarg)[0] ;
            padAnRight = padAnLeft ;
            padAnTop = padAnLeft ;
            padAnBottom = padAnLeft ;
            break ;
          case 4:
            padAnTop = (int)mxGetPr(optarg)[0] ;
            padAnBottom = (int)mxGetPr(optarg)[1] ;
            padAnLeft = (int)mxGetPr(optarg)[2] ;
            padAnRight = (int)mxGetPr(optarg)[3] ;
            break ;
          default:
            vlmxError(VLMXE_IllegalArgument, "PADANGULAR has neither one nor four elements.") ;
        }
        break ;

      case opt_no_der_data :
        computeDerData = false ;
        break ;

      case opt_no_der_filters :
        computeDerFilters = false ;
        break ;

      case opt_no_der_biases :
        computederBiases = false ;
        break ;

      case opt_no_cudnn :
#if ENABLE_CUDNN
        context.getCudaHelper().setCudnnEnabled(false) ;
#endif
        break ;

      case opt_cudnn :
#if ENABLE_CUDNN
        context.getCudaHelper().setCudnnEnabled(true) ;
#endif
        break ;

      case opt_cudnn_workspace_limit :
      {
#if ENABLE_CUDNN
        double x ;
        if (!vlmxIsScalar(optarg) || (x = mxGetScalar(optarg)) < 0) {
          vlmxError(VLMXE_IllegalArgument, "CudnnWorkSpaceLimit is not a non-negative scalar.") ;
        }
        context.getCudaHelper().setCudnnConvolutionFwdPreference
        ((x==mxGetInf() ?
          HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST :
          HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT),
         (size_t)x) ;
        context.getCudaHelper().setCudnnConvolutionBwdFilterPreference
        ((x==mxGetInf() ?
          HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST :
          HIPDNN_CONVOLUTION_BWD_FILTER_SPECIFY_WORKSPACE_LIMIT),
         (size_t)x) ;
        context.getCudaHelper().setCudnnConvolutionBwdDataPreference
        ((x==mxGetInf() ?
          HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST :
          HIPDNN_CONVOLUTION_BWD_DATA_SPECIFY_WORKSPACE_LIMIT),
         (size_t)x) ;
        break ;
#endif
      }

      default: break ;
    }
  }

  vl::MexTensor data(context) ;
  vl::MexTensor filters(context) ;
  vl::MexTensor biases(context) ;
  vl::MexTensor derOutput(context) ;

  data.init(in[IN_DATA]) ;
  data.reshape(6) ;

  filters.init(in[IN_FILTERS]) ;
  filters.reshape(6) ;

  biases.init(in[IN_BIASES]) ;

  if (backMode) {
    derOutput.init(in[IN_DEROUTPUT]) ;
    derOutput.reshape(6) ;
  }

  hasFilters = !filters.isEmpty() ;
  hasBiases = !biases.isEmpty() ;

  /* check for GPU/data class consistency */
  if (hasFilters && ! vl::areCompatible(data, filters)) {
    vlmxError(VLMXE_IllegalArgument, "DATA and FILTERS do not have compatible formats.") ;
  }
  if (hasBiases && ! vl::areCompatible(data, biases)) {
    vlmxError(VLMXE_IllegalArgument, "DATA and BIASES do not have compatible formats.") ;
  }
  if (backMode && ! vl::areCompatible(data, derOutput)) {
    vlmxError(VLMXE_IllegalArgument, "DATA and DEROUTPUT do not have compatible formats.") ;
  }

  /* basic argument checks */
  if (strideX < 1 || strideY < 1 || strideAnX < 1 || strideAnY < 1) {
    vlmxError(VLMXE_IllegalArgument, "At least one element of STRIDE or STRIDEANGULAR is smaller than one.") ;
  }
  if (padLeft < 0 ||
      padRight < 0 ||
      padTop < 0 ||
      padBottom < 0 ||
      padAnLeft < 0 ||
      padAnRight < 0 ||
      padAnTop < 0 ||
      padAnBottom < 0) {
    vlmxError(VLMXE_IllegalArgument, "An element of PAD or PADANGULAR is negative.") ;
  }

  /* Get the filter shape */
  vl::TensorShape filtersShape(filters) ;
  int equivalentNumFilters ;
  if (hasFilters) {
    if (filtersShape.getDimension(0) == 0 || filtersShape.getDimension(1) == 0 || filtersShape.getDimension(2) == 0
|| filtersShape.getDimension(3) == 0 || filtersShape.getDimension(4) == 0) {
      vlmxError(VLMXE_IllegalArgument, "A dimension of FILTERS is void.") ;
    }
    if (data.getDimension(0) + (padTop+padBottom) < filters.getDimension(0) ||
        data.getDimension(1) + (padLeft+padRight) < (filters.getDimension(1))) {
      vlmxError(VLMXE_IllegalArgument, "FILTERS are larger than the DATA (including padding).") ;
    }
    if (data.getDimension(2) + (padAnTop+padAnBottom) < filters.getDimension(2) ||
        data.getDimension(3) + (padAnLeft+padAnRight) < (filters.getDimension(3))) {
      vlmxError(VLMXE_IllegalArgument, "ANGULARFILTERS are larger than the DATA (including padding).") ;
    }
    /* grouped filters */
    numFilterGroups = data.getDimension(4) / filters.getDimension(4) ;
    if (numFilterGroups * filters.getDimension(4) != data.getDimension(4)) {
      vlmxError(VLMXE_IllegalArgument, "The FILTERS depth does not divide the DATA depth.") ;
    }
    if (filters.getDimension(5) % numFilterGroups != 0) {
      vlmxError(VLMXE_IllegalArgument, "The number of filter groups does not divide the number of filters.") ;
    }
    equivalentNumFilters = filters.getDimension(5) ;
  } else {
    vlmxError(VLMXE_IllegalArgument, "There is no filters specified.") ;
  }

  /* Get the output shape */
  int kernelExtentX = filtersShape.getDimension(1);
  int kernelExtentY = filtersShape.getDimension(0);
  int kernelExtentAnX = filtersShape.getDimension(3);
  int kernelExtentAnY = filtersShape.getDimension(2);
  
  
  vl::TensorShape outputShape((data.getDimension(0) + (padTop+padBottom) - kernelExtentY)/strideY + 1,
                                (data.getDimension(1)  + (padLeft+padRight) - kernelExtentX)/strideX + 1,
                                 (data.getDimension(2) + (padAnTop+padAnBottom) - kernelExtentAnY)/strideAnY + 1,
                                  (data.getDimension(3)  + (padAnLeft+padAnRight) - kernelExtentAnX)/strideAnX + 1,
                                equivalentNumFilters,
                                data.getDimension(5)) ;
 
  
  if (backMode && (derOutput != outputShape)) {
    vlmxError(VLMXE_IllegalArgument, "DEROUTPUT dimensions are incompatible with X and FILTERS.") ;
  }

  /* Check the biases sizes */
  if (hasBiases) {
    if (biases.getNumElements() != filtersShape.getDimension(5)) {
      vlmxError(VLMXE_IllegalArgument, "The number of elements of BIASES is not the same as the number of filters.") ;
    }
  }

  /* create output buffers */
  vl::DeviceType deviceType = data.getDeviceType() ;
  vl::DataType dataType = data.getDataType() ;
  vl::MexTensor output(context) ;
  vl::MexTensor derData(context) ;
  vl::MexTensor derFilters(context) ;
  vl::MexTensor derBiases(context) ;

  if (!backMode) {
    output.init(deviceType, dataType, outputShape) ;
  } else {
    if (computeDerData) {
      derData.init(deviceType, dataType, data.getShape()) ;
    }
    if (computeDerFilters && hasFilters) {
      derFilters.init(deviceType, dataType, filters.getShape()) ;
    }
    if (computederBiases && hasBiases) {
      derBiases.init(deviceType, dataType, biases.getShape()) ;
    }
  }

  if (verbosity > 0) {
    mexPrintf("vl_nnconv6D: %s; %s", backMode?"backward":"forward", (data.getDeviceType()==vl::VLDT_GPU) ? "GPU" : "CPU") ;
    if (data.getDeviceType() == vl::VLDT_GPU) {
#if ENABLE_CUDNN
      mexPrintf("; %s\n", context.getCudaHelper().getCudnnEnabled() ? "cuDNN" : "cuBLAS") ;
#else
      mexPrintf("; cuBLAS\n") ;
#endif
    } else {
      mexPrintf("; BLAS\n") ;
    }
    mexPrintf("vl_nnconv6D: stride: [%d %d], stride angular: [%d %d]\n"
              "vl_nnconv6D: pad: [%d %d %d %d], pad angular: [%d %d %d %d]\n"
              "vl_nnconv6D: num filter groups: %d\n",
              strideY, strideX,
              strideAnY, strideAnX,
              padTop, padBottom, padLeft, padRight,
              padAnTop, padAnBottom, padAnLeft, padAnRight,
              numFilterGroups) ;
    vl::print("vl_nnconv6D: data: ", data) ;
    if (hasFilters) { vl::print("vl_nnconv6D: filters: ", filters) ; }
    if (hasBiases) { vl::print("vl_nnconv6D: biases: ", biases) ; }
    if (backMode) {
      vl::print("vl_nnconv6D: derOutput: ", derOutput) ;
      vl::print("vl_nnconv6D: derData: ", derData) ;
      if (hasFilters) { vl::print("vl_nnconv6D: derFilters: ", derFilters) ; }
      if (hasBiases) { vl::print("vl_nnconv6D: derBiases: ", derBiases) ; }
    } else {
      vl::print("vl_nnconv6D: output: ", output) ;
    }
  }

  /* -------------------------------------------------------------- */
  /*                                                    Do the work */
  /* -------------------------------------------------------------- */

  vl::ErrorCode error ;

  /* regular case */
  if (!backMode) {
    error = vl::nnconv6D_forward(context,
                               output, 0,
                               data, 1,
                               filters,
                               biases,
                               strideY, strideX,
                               strideAnY, strideAnX,
                               padTop, padBottom, padLeft, padRight,
                               padAnTop, padAnBottom, padAnLeft, padAnRight) ;
  } else {
    error = vl::nnconv6D_backward(context,
                                derData,
                                derFilters,
                                derBiases,
                                data,
                                filters,
                                derOutput,
                                strideY, strideX,
                                strideAnY, strideAnX,
                                padTop, padBottom, padLeft, padRight,
                                padAnTop, padAnBottom, padAnLeft, padAnRight) ;
  }

doneok:
  if (verbosity > 0) {
#if ENABLE_CUDNN
    if (context.getCudaHelper().getCudnnEnabled()) {
      mexPrintf("vl_nnconv6D: cuDNN workspace used: "
                "fwd %.6g MB"
                ", bwd filter %.6g MB"
                ", bwd data %.6g MB\n",
                (double)context.getCudaHelper().getCudnnConvolutionFwdWorkSpaceUsed() / (1024*1024),
                (double)context.getCudaHelper().getCudnnConvolutionBwdFilterWorkSpaceUsed() / (1024*1024),
                (double)context.getCudaHelper().getCudnnConvolutionBwdDataWorkSpaceUsed() / (1024*1024)) ;
    }
#endif
  }

  /* -------------------------------------------------------------- */
  /*                                                        Cleanup */
  /* -------------------------------------------------------------- */

  if (error != vl::VLE_Success) {
    vlmxError(VLMXE_IllegalArgument, context.getLastErrorMessage().c_str()) ;
  }
  if (backMode) {
    mxClassID classID ;
    switch (derOutput.getDataType()) {
      case vl::VLDT_Float: classID = mxSINGLE_CLASS ; break ;
      case vl::VLDT_Double: classID = mxDOUBLE_CLASS ; break ;
      default: abort() ;
    }
    out[OUT_RESULT] = (computeDerData) ? derData.relinquish() : mxCreateNumericMatrix(0,0,classID,mxREAL) ;
    out[OUT_DERFILTERS] = (computeDerFilters & hasFilters)? derFilters.relinquish() : mxCreateNumericMatrix(0,0,classID,mxREAL) ;
    out[OUT_DERBIASES] = (computederBiases & hasBiases) ? derBiases.relinquish() : mxCreateNumericMatrix(0,0,classID,mxREAL) ;
  } else {
    out[OUT_RESULT] = output.relinquish() ;
  }
}
