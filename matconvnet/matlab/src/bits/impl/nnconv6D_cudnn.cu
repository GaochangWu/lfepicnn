#include "hip/hip_runtime.h"
// @file nnconv_cudnn.cu
// @brief Convolution block CuDNN-based implementation.
// @author Andrea Vedaldi

/*
Copyright (C) 2015-16 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#if !defined(ENABLE_GPU) | !defined(ENABLE_CUDNN)
#error "nnconv6D_cudnn.hpp cannot be compiled without GPU and CUDNN support."
#endif

#include "nnconv6D_cudnn.hpp"
#include "cudnnhelper.hpp"
#include "../datacu.hpp"
#include <assert.h>
#include <algorithm>
#include <iostream>

using namespace vl ;

#define CHECK(x) \
{ \
cudnnError = x ; \
if (cudnnError != HIPDNN_STATUS_SUCCESS) { \
error = context.setError(context.getCudaHelper().catchCudnnError(cudnnError, \
STRINGIZE(__FILE__) ":" STRINGIZE(__LINE__))) ; \
goto done ; \
} }

/* ---------------------------------------------------------------- */
/*                                             nnconv_forward_cudnn */
/* ---------------------------------------------------------------- */

namespace vl { namespace impl {

  template<vl::DataType dataType>
  vl::ErrorCode
  vl::impl::nnconv6D_cudnn<dataType>::forward(Context& context,
                                            Tensor output, double outputMult,
                                            Tensor data, double dataMult,
                                            Tensor filters,
                                            Tensor biases,
                                            int strideY, int strideX,
                                            int strideAnY, int strideAnX,
                                            int padTop, int padBottom,
                                            int padLeft, int padRight,
                                            int padAnTop, int padAnBottom,
                                            int padAnLeft, int padAnRight)
  {
    assert(output) ;
    assert(data) ;
    assert(filters) ;

    typedef typename DataTypeTraits<dataType>::type type ;

    hipdnnTensorDescriptor_t outputDesc, biasesDesc, dataDesc ;
    hipdnnFilterDescriptor_t filtersDesc ;
    hipdnnConvolutionDescriptor_t convDesc ;
    bool outputDescInitialized = false ;
    bool biasesDescInitialized = false ;
    bool dataDescInitialized = false ;
    bool filtersDescInitialized = false ;
    bool convDescInitialized = false ;

    void* workSpace = NULL ;

    int numGroups = data.getDimension(4) / filters.getDimension(4) ;
    int numFiltersPerGroup = filters.getSize() / numGroups ;

    if (padLeft != padRight) return vl::VLE_Unsupported ;
    if (padTop != padBottom) return vl::VLE_Unsupported ;
    if (padAnLeft != padAnRight) return vl::VLE_Unsupported ;
    if (padAnTop != padAnBottom) return vl::VLE_Unsupported ;
    if (filters.getDimension(0) > data.getDimension(0)) return vl::VLE_Unsupported ;
    if (filters.getDimension(1) > data.getDimension(1)) return vl::VLE_Unsupported ;
    if (filters.getDimension(2) > data.getDimension(2)) return vl::VLE_Unsupported ;
    if (filters.getDimension(3) > data.getDimension(3)) return vl::VLE_Unsupported ;

    hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
    vl::ErrorCode error = vl::VLE_Success ;
    hipdnnHandle_t handle ;

    // Get CuDNN
    CHECK(context.getCudaHelper().getCudnnHandle(&handle)) ;

    // Get tensor descripotrs
    {
    CHECK(hipdnnCreateTensorDescriptor(&outputDesc)) ;
    outputDescInitialized = true ;
    
    int out_n = output.getDimension(5);
    int out_c = output.getDimension(4);
    int out_w = output.getDimension(0);
    int out_h = output.getDimension(1);
    int out_an_w = output.getDimension(2);
    int out_an_h = output.getDimension(3);
    int out_dims [6] = {out_n, out_c, out_w, out_h, out_an_w, out_an_h};
    int out_strides [6] = {out_c*out_w*out_h*out_an_w*out_an_h, out_w*out_h*out_an_w*out_an_h,
                       out_h*out_an_w*out_an_h, out_an_w*out_an_h, out_an_h, 1};

    CHECK(hipdnnSetTensorNdDescriptor(outputDesc,
                                       DataTypeToCudnn<dataType>::id ,
                                       6 ,
                                       out_dims,
                                       out_strides)) ;
    }
    
    {
    CHECK(hipdnnCreateTensorDescriptor(&dataDesc)) ;
    dataDescInitialized = true ;

    int data_n = data.getDimension(5);
    int data_c = data.getDimension(4);
    int data_w = data.getDimension(0);
    int data_h = data.getDimension(1);
    int data_an_w = data.getDimension(2);
    int data_an_h = data.getDimension(3);
    int data_dims [6] = {data_n, data_c, data_w, data_h, data_an_w, data_an_h};
    int data_strides [6] = {data_c*data_w*data_h*data_an_w*data_an_h, data_w*data_h*data_an_w*data_an_h,
                       data_h*data_an_w*data_an_h, data_an_w*data_an_h, data_an_h, 1};

    CHECK(hipdnnSetTensorNdDescriptor(dataDesc,
                                       DataTypeToCudnn<dataType>::id,
                                       6,
                                       data_dims,
                                       data_strides)) ;
    }

    {
    CHECK(hipdnnCreateFilterDescriptor(&filtersDesc)) ;
    filtersDescInitialized = true ;

    int filter_n = filters.getDimension(5);
    int filter_c = filters.getDimension(4);
    int filter_w = filters.getDimension(0);
    int filter_h = filters.getDimension(1);
    int filter_an_w = filters.getDimension(2);
    int filter_an_h = filters.getDimension(3);
    int filter_dims [6] = {filter_n, filter_c, filter_w, filter_h, filter_an_w, filter_an_h};    

    CHECK(hipdnnSetFilterNdDescriptor(filtersDesc,
                                     DataTypeToCudnn<dataType>::id,
                                     IF_CUDNN_GE5(HIPDNN_TENSOR_NCHW COMMA)
                                     6,
                                     filter_dims)) ;
    }

    if (biases) {
      CHECK(hipdnnCreateTensorDescriptor(&biasesDesc)) ;
      biasesDescInitialized = true ;

      int bias_c = biases.getNumElements() / numGroups;
      int bias_dims [6] = {1,bias_c,1,1,1,1};
      int bias_strides [6] = { bias_c, 1,1,1,1,1};
      
      CHECK(hipdnnSetTensorNdDescriptor(biasesDesc,
                                       DataTypeToCudnn<dataType>::id ,
                                       6,
                                       bias_dims,
                                       bias_strides)) ;
    }

    // Get convolution descriptor
    {
    CHECK(hipdnnCreateConvolutionDescriptor(&convDesc)) ;
    convDescInitialized = true ;

    int conv_pad [4] = {padLeft, padTop,padAnLeft, padAnTop};
    int conv_stride [4] = {strideX, strideY, strideAnX, strideAnY};
    int conv_up [4] = {1,1,1,1};
    CHECK(hipdnnSetConvolutionNdDescriptor(convDesc,
                                          4,
                                          conv_pad,
                                          conv_stride,
                                          conv_up, // upscale
                                          HIPDNN_CROSS_CORRELATION,
                                          DataTypeToCudnn<dataType>::id)) ;
    }
    // Sanity check

#if 1
    {
      int output_dims [6];
      cudnnGetConvolutionNdForwardOutputDim(convDesc,
                                            dataDesc,
                                            filtersDesc,
                                            6,
                                            output_dims) ;
      bool sane =
      output.getDimension(5) == output_dims[0] &&
      numFiltersPerGroup == output_dims[1] &&
      output.getDimension(0) == output_dims[2] &&
      output.getDimension(1) == output_dims[3] &&
      output.getDimension(2) == output_dims[4] &&
      output.getDimension(3) == output_dims[5] ;
      assert(sane) ;
    }
#endif
    context.getCudaHelper().getCudnnEnabled();
    context.getCudaHelper().cudnnConvolutionFwdWorkSpaceUsed = 0 ;
    context.getCudaHelper().cudnnConvolutionBwdFilterWorkSpaceUsed = 0 ;
    context.getCudaHelper().cudnnConvolutionBwdDataWorkSpaceUsed = 0 ;
    
    
    if (!context.getCudaHelper().cudnnConvolutionFwdSpecificAlgo) {
      // Determine algorithm automatically
      CHECK(hipdnnGetConvolutionForwardAlgorithm(handle,
                                                dataDesc,
                                                filtersDesc,
                                                convDesc,
                                                outputDesc,
                                                context.getCudaHelper().cudnnConvolutionFwdPreference,
                                                context.getCudaHelper().cudnnConvolutionFwdWorkSpaceLimit,
                                                &context.getCudaHelper().cudnnConvolutionFwdAlgo)) ;
    }

    // Get workspace size
    CHECK(hipdnnGetConvolutionForwardWorkspaceSize(handle,
                                                  dataDesc,
                                                  filtersDesc,
                                                  convDesc,
                                                  outputDesc,
                                                  context.getCudaHelper().cudnnConvolutionFwdAlgo,
                                                  &context.getCudaHelper().cudnnConvolutionFwdWorkSpaceUsed)) ;

    // Get workspace
    if (context.getCudaHelper().cudnnConvolutionFwdWorkSpaceUsed > 0) {
      workSpace = context.getWorkspace(vl::VLDT_GPU, context.getCudaHelper().cudnnConvolutionFwdWorkSpaceUsed) ;
      if (workSpace == NULL) {
        error = context.getLastError() ;
        goto done ;
      }
    }

    // Perform convolution for each filter group
    for (int g = 0  ; g < numGroups ; ++g) {
      ptrdiff_t dataGrpOffset = (data.getHeight() * data.getWidth() * filters.getDepth()) *  g ;
      ptrdiff_t filtersGrpOffset = (filters.getHeight() * filters.getWidth() * filters.getDepth()) * numFiltersPerGroup * g ;
      ptrdiff_t outputGrpOffset = (output.getHeight() * output.getWidth() * numFiltersPerGroup) * g ;
      ptrdiff_t biasesGrpOffset = numFiltersPerGroup * g ;

      type alpha = dataMult ;
      type beta = outputMult ;
      CHECK(hipdnnConvolutionForward(handle,
                                    &alpha,
                                    dataDesc, (type const*)data.getMemory() + dataGrpOffset,
                                    filtersDesc, (type const*)filters.getMemory() + filtersGrpOffset,
                                    convDesc,
                                    context.getCudaHelper().cudnnConvolutionFwdAlgo,
                                    workSpace, context.getCudaHelper().cudnnConvolutionFwdWorkSpaceUsed,
                                    &beta,
                                    outputDesc, (type*)output.getMemory() + outputGrpOffset)) ;

      if (biases) {
        type alpha = 1.0f ;
        type beta = 1.0f ;
#if (CUDNN_VERSION < 4000)
        CHECK(hipdnnAddTensor(handle,
                             CUDNN_ADD_SAME_C,
                             &alpha,
                             biasesDesc, (type const*)biases.getMemory() + biasesGrpOffset,
                             &beta,
                             outputDesc, (type*)output.getMemory() + outputGrpOffset)) ;
#else
        CHECK(hipdnnAddTensor(handle,
                             &alpha,
                             biasesDesc, (type const*)biases.getMemory() + biasesGrpOffset,
                             &beta,
                             outputDesc, (type*)output.getMemory() + outputGrpOffset)) ;
#endif
      }
    }

    /* cleanup */
  done:
    if (convDescInitialized) { hipdnnDestroyConvolutionDescriptor(convDesc) ; }
    if (filtersDescInitialized) { hipdnnDestroyFilterDescriptor(filtersDesc) ; }
    if (dataDescInitialized) { hipdnnDestroyTensorDescriptor(dataDesc) ; }
    if (biasesDescInitialized) { hipdnnDestroyTensorDescriptor(biasesDesc) ; }
    if (outputDescInitialized) { hipdnnDestroyTensorDescriptor(outputDesc) ; }
    return context.passError(error, __func__) ;
  }

  /* ---------------------------------------------------------------- */
  /*                                            nnconv_backward_cudnn */
  /* ---------------------------------------------------------------- */

  template<vl::DataType dataType>
  vl::ErrorCode
  vl::impl::nnconv6D_cudnn<dataType>::backward(Context& context,
                                             Tensor derData,
                                             Tensor derFilters,
                                             Tensor derBiases,
                                             Tensor data,
                                             Tensor filters,
                                             Tensor derOutput,
                                             int strideY, int strideX,
                                             int strideAnY, int strideAnX,
                                             int padTop, int padBottom,
                                             int padLeft, int padRight,
                                             int padAnTop, int padAnBottom,
                                             int padAnLeft, int padAnRight)
  {
    typedef typename DataTypeTraits<dataType>::type type ;

    /* no derDataDesc needed as same as dataDesc */
    hipdnnTensorDescriptor_t dataDesc, derBiasesDesc, derOutputDesc ;
    hipdnnFilterDescriptor_t filtersDesc ;
    hipdnnConvolutionDescriptor_t convDesc ;
    bool dataDescInitialized = false ;
    bool derBiasesDescInitialized = false ;
    bool derOutputDescInitialized = false ;
    bool filtersDescInitialized = false ;
    bool convDescInitialized = false ;

#if (CUDNN_VERSION >= 3000)
    void* workSpace = NULL ;
    size_t workSpaceSize = 0 ;
#endif

    ptrdiff_t numGroups = 1 ;
    ptrdiff_t numFiltersPerGroup = 0 ;
    ptrdiff_t filtersVolume = 0 ;

    if (padLeft != padRight) return vl::VLE_Unsupported ;
    if (padTop != padBottom) return vl::VLE_Unsupported ;
    if (padAnLeft != padAnRight) return vl::VLE_Unsupported ;
    if (padAnTop != padAnBottom) return vl::VLE_Unsupported ;

    hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
    vl::ErrorCode error = vl::VLE_Success ;
    hipdnnHandle_t handle ;

    // Get CuDNN
    CHECK(context.getCudaHelper().getCudnnHandle(&handle)) ;

    // Get the dimensions of the tensrors involved
    // If derData is specified (hence comptued as output), use this
    // tensor as a basis to compute such dimensions, otherwise use derFilters.

    if (derData) {
      assert(filters) ;
      numGroups = derData.getDimension(4) / filters.getDimension(4) ;
      numFiltersPerGroup = filters.getDimension(5) / numGroups ;
      filtersVolume = filters.getDimension(0) * filters.getDimension(1) * filters.getDimension(2) 
                      * filters.getDimension(3) * filters.getDimension(4) ;

      
      CHECK(hipdnnCreateTensorDescriptor(&dataDesc)) ;
      dataDescInitialized = true ;

      int der_data_n = derData.getDimension(5);
      int der_data_c = derData.getDimension(4);
      int der_data_w = derData.getDimension(0);
      int der_data_h = derData.getDimension(1);
      int der_data_an_w = derData.getDimension(2);
      int der_data_an_h = derData.getDimension(3);
      int der_data_dims [6] = {der_data_n, der_data_c, der_data_w, der_data_h, der_data_an_w, der_data_an_h};
      int der_data_strides [6] = {der_data_c*der_data_w*der_data_h*der_data_an_w*der_data_an_h, der_data_w*der_data_h*der_data_an_w*der_data_an_h,
                       der_data_h*der_data_an_w*der_data_an_h, der_data_an_w*der_data_an_h, der_data_an_h, 1};

      CHECK(hipdnnSetTensorNdDescriptor(dataDesc,
                                         DataTypeToCudnn<dataType>::id ,
                                         6,
                                         der_data_dims,
                                         der_data_strides)) ;

      

      CHECK(hipdnnCreateFilterDescriptor(&filtersDesc)) ;
      filtersDescInitialized = true ;

      int filter_n = filters.getDimension(5);
      int filter_c = filters.getDimension(4);
      int filter_w = filters.getDimension(0);
      int filter_h = filters.getDimension(1);
      int filter_an_w = filters.getDimension(2);
      int filter_an_h = filters.getDimension(3);
      int filter_dims [6] = {filter_n, filter_c, filter_w, filter_h, filter_an_w, filter_an_h};

      CHECK(hipdnnSetFilterNdDescriptor(filtersDesc,
                                       DataTypeToCudnn<dataType>::id ,
                                       IF_CUDNN_GE5(HIPDNN_TENSOR_NCHW COMMA)
                                       6,
                                       filter_dims)) ;
    } else if (derFilters) {
      assert(data) ;
      numGroups = data.getDimension(4) / derFilters.getDimension(4) ;
      numFiltersPerGroup = derFilters.getDimension(5) / numGroups ;
      filtersVolume = derFilters.getDimension(0) * derFilters.getDimension(1) * derFilters.getDimension(2) 
                      * derFilters.getDimension(3) * derFilters.getDimension(4) ;

      CHECK(hipdnnCreateTensorDescriptor(&dataDesc)) ;
      dataDescInitialized = true ;
      
      int data_n = data.getDimension(5);
      int data_c = data.getDimension(4);
      int data_w = data.getDimension(0);
      int data_h = data.getDimension(1);
      int data_an_w = data.getDimension(2);
      int data_an_h = data.getDimension(3);
      int data_dims [6] = {data_n, data_c, data_w, data_h, data_an_w, data_an_h};
      int data_strides [6] = {data_c*data_w*data_h*data_an_w*data_an_h, data_w*data_h*data_an_w*data_an_h,
                       data_h*data_an_w*data_an_h, data_an_w*data_an_h, data_an_h, 1};

      CHECK(hipdnnSetTensorNdDescriptor(dataDesc,
                                         DataTypeToCudnn<dataType>::id ,
                                         6,
                                         data_dims,
                                         data_strides)) ;

      CHECK(hipdnnCreateFilterDescriptor(&filtersDesc)) ;
      filtersDescInitialized = true ;

      int derFilter_n = derFilters.getDimension(5);
      int derFilter_c = derFilters.getDimension(4);
      int derFilter_w = derFilters.getDimension(0);
      int derFilter_h = derFilters.getDimension(1);
      int derFilter_an_w = derFilters.getDimension(2);
      int derFilter_an_h = derFilters.getDimension(3);
      int derFilter_dims [6] = {derFilter_n, derFilter_c, derFilter_w, derFilter_h, derFilter_an_w, derFilter_an_h};

      CHECK(hipdnnSetFilterNdDescriptor(filtersDesc,
                                       DataTypeToCudnn<dataType>::id ,
                                       IF_CUDNN_GE5(HIPDNN_TENSOR_NCHW COMMA)
                                       6,
                                       derFilter_dims)) ;
    }

    {       
    CHECK(hipdnnCreateConvolutionDescriptor(&convDesc)) ;
    convDescInitialized = true ;
    int conv_pad [4] = {padLeft, padTop,padAnLeft, padAnTop};
    int conv_stride [4] = {strideX, strideY, strideAnX, strideAnY};
    int conv_up [4] = {1,1,1,1};
    CHECK(hipdnnSetConvolutionNdDescriptor(convDesc,
                                          4,
                                          conv_pad,
                                          conv_stride,
                                          conv_up, // upscale
                                          HIPDNN_CROSS_CORRELATION,
                                          DataTypeToCudnn<dataType>::id)) ;
    }

    // Must have derOutput for all derivatives
    {
    assert(derOutput) ;
    CHECK(hipdnnCreateTensorDescriptor(&derOutputDesc)) ;
    derOutputDescInitialized = true ;

    int der_out_n = derOutput.getDimension(5);
    int der_out_c = derOutput.getDimension(4);
    int der_out_w = derOutput.getDimension(0);
    int der_out_h = derOutput.getDimension(1);
    int der_out_an_w = derOutput.getDimension(2);
    int der_out_an_h = derOutput.getDimension(3);
    int der_out_dims [6] = {der_out_n, der_out_c, der_out_w, der_out_h, der_out_an_w, der_out_an_h};
    int der_out_strides [6] = {der_out_c*der_out_w*der_out_h*der_out_an_w*der_out_an_h, der_out_w*der_out_h*der_out_an_w*der_out_an_h,
                       der_out_h*der_out_an_w*der_out_an_h, der_out_an_w*der_out_an_h, der_out_an_h, 1};

    CHECK(hipdnnSetTensorNdDescriptor(derOutputDesc,
                                       DataTypeToCudnn<dataType>::id ,
                                       6, 
                                       der_out_dims,
                                       der_out_strides)) ;
    }
    // for derivatives w.r.t. bias
    if (derBiases) {
      CHECK(hipdnnCreateTensorDescriptor(&derBiasesDesc)) ;
      derBiasesDescInitialized = true ;

      int der_bias_c = derBiases.getNumElements() / numGroups;
      int der_bias_dims [6] = {1,der_bias_c,1,1,1,1};
      int der_bias_strides [6] = { der_bias_c, 1,1,1,1,1};
      
      CHECK(hipdnnSetTensorNdDescriptor(derBiasesDesc,
                                       DataTypeToCudnn<dataType>::id ,
                                       6,
                                       der_bias_dims,
                                       der_bias_strides)) ;
    }


    context.getCudaHelper().cudnnConvolutionFwdWorkSpaceUsed = 0 ;
    context.getCudaHelper().cudnnConvolutionBwdFilterWorkSpaceUsed = 0 ;
    context.getCudaHelper().cudnnConvolutionBwdDataWorkSpaceUsed = 0 ;

#if (CUDNN_VERSION >= 3000)

    if (derFilters) {
      // Get filter derivatives algorithm
      CHECK(hipdnnGetConvolutionBackwardFilterAlgorithm
            (handle,
             dataDesc,
             derOutputDesc,
             convDesc,
             filtersDesc,
             context.getCudaHelper().cudnnConvolutionBwdFilterPreference,
             context.getCudaHelper().cudnnConvolutionBwdFilterWorkSpaceLimit,
             &context.getCudaHelper().cudnnConvolutionBwdFilterAlgo)) ;

      // Get workspace size
      CHECK(hipdnnGetConvolutionBackwardFilterWorkspaceSize
            (handle,
             dataDesc,
             derOutputDesc,
             convDesc,
             filtersDesc,
             context.getCudaHelper().cudnnConvolutionBwdFilterAlgo,
             &context.getCudaHelper().cudnnConvolutionBwdFilterWorkSpaceUsed)) ;
      workSpaceSize = std::max(workSpaceSize, context.getCudaHelper().cudnnConvolutionBwdFilterWorkSpaceUsed) ;
    }

    if (derData) {
      // Get data derivatives
      CHECK(hipdnnGetConvolutionBackwardDataAlgorithm
            (handle,
             filtersDesc,
             derOutputDesc,
             convDesc,
             dataDesc,
             context.getCudaHelper().cudnnConvolutionBwdDataPreference,
             context.getCudaHelper().cudnnConvolutionBwdDataWorkSpaceLimit,
             &context.getCudaHelper().cudnnConvolutionBwdDataAlgo)) ;

      // Get workspace size
      CHECK(hipdnnGetConvolutionBackwardDataWorkspaceSize
            (handle,
             filtersDesc,
             derOutputDesc,
             convDesc,
             dataDesc,
             context.getCudaHelper().cudnnConvolutionBwdDataAlgo,
             &context.getCudaHelper().cudnnConvolutionBwdDataWorkSpaceUsed)) ;
      workSpaceSize = std::max(workSpaceSize, context.getCudaHelper().cudnnConvolutionBwdDataWorkSpaceUsed) ;
    }

    // Get workspace
    if (workSpaceSize > 0) {
      workSpace = context.getWorkspace(vl::VLDT_GPU, workSpaceSize) ;
      if (workSpace == NULL) {
        error = context.getLastError() ;
        goto done ;
      }
    }
#endif

    // Perform backward convolution for each filter group
    for (int g = 0  ; g < numGroups ; ++g) {
      ptrdiff_t filtersGrpOffset = filtersVolume * numFiltersPerGroup  * g ;
      ptrdiff_t derOutputGrpOffset = (derOutput.getHeight() * derOutput.getWidth() * numFiltersPerGroup) * g ;

      if (derBiases) {
        ptrdiff_t derBiasesGrpOffset = numFiltersPerGroup * g ;
        type alpha = 1 ;
        type beta = 0 ;
        CHECK(hipdnnConvolutionBackwardBias
              (handle,
               &alpha,
               derOutputDesc, (type const*)derOutput.getMemory() + derOutputGrpOffset,
               &beta,
               derBiasesDesc, (type*)derBiases.getMemory() + derBiasesGrpOffset)) ;
      }

      if (derFilters) {
        ptrdiff_t dataGrpOffset = (data.getHeight() * data.getWidth() * derFilters.getDepth()) *  g ;
        type alpha = 1 ;
        type beta = 0 ;
#if (CUDNN_VERSION >= 3000)
        CHECK(
              IF_CUDNN_GE4(hipdnnConvolutionBackwardFilter)
              IF_CUDNN_GE3_LT4(cudnnConvolutionBackwardFilter_v3)
              (handle,
               &alpha,
               dataDesc, (type const*)data.getMemory() + dataGrpOffset,
               derOutputDesc, (type const*)derOutput.getMemory() + derOutputGrpOffset,
               convDesc,
               context.getCudaHelper().cudnnConvolutionBwdFilterAlgo,
               workSpace, workSpaceSize,
               &beta,
               filtersDesc, (type*)derFilters.getMemory() + filtersGrpOffset)) ;
#else
        CHECK(hipdnnConvolutionBackwardFilter
              (handle,
               &alpha,
               dataDesc, (type const*)data.getMemory() + dataGrpOffset,
               derOutputDesc, (type const*)derOutput.getMemory() + derOutputGrpOffset,
               convDesc,
               &beta,
               filtersDesc, (type*)derFilters.getMemory() + filtersGrpOffset)) ;
#endif
      }

      if (derData) {
        ptrdiff_t dataGrpOffset = (derData.getHeight() * derData.getWidth() * filters.getDepth()) *  g ;
        type alpha = 1 ;
        type beta = 0 ;

#if (CUDNN_VERSION >= 3000)
        CHECK(
              IF_CUDNN_GE4(hipdnnConvolutionBackwardData)
              IF_CUDNN_GE3_LT4(cudnnConvolutionBackwardData_v3)
              (handle,
               &alpha,
               filtersDesc, (type const*)filters.getMemory() + filtersGrpOffset,
               derOutputDesc, (type const*)derOutput.getMemory() + derOutputGrpOffset,
               convDesc,
               context.getCudaHelper().cudnnConvolutionBwdDataAlgo,
               workSpace, workSpaceSize,
               &beta,
               dataDesc, (type*)derData.getMemory() + dataGrpOffset)) ;
#else
        CHECK(hipdnnConvolutionBackwardData
              (handle,
               &alpha,
               filtersDesc, filters.getMemory() + filtersGrpOffset,
               derOutputDesc, derOutput.getMemory() + derOutputGrpOffset,
               convDesc,
               &beta,
               dataDesc, derData.getMemory() + dataGrpOffset)) ;
#endif
      }
    }

  done:
    if (convDescInitialized) { hipdnnDestroyConvolutionDescriptor(convDesc) ; }
    if (filtersDescInitialized) { hipdnnDestroyFilterDescriptor(filtersDesc) ; }
    if (derOutputDescInitialized) { hipdnnDestroyTensorDescriptor(derOutputDesc) ; }
    if (derBiasesDescInitialized) { hipdnnDestroyTensorDescriptor(derBiasesDesc) ; }
    if (dataDescInitialized) { hipdnnDestroyTensorDescriptor(dataDesc) ; }
    return context.passError(error, __func__) ;
  }

} }

// Instantiations
template struct vl::impl::nnconv6D_cudnn<vl::VLDT_Float> ;

#ifdef ENABLE_DOUBLE
template struct vl::impl::nnconv6D_cudnn<vl::VLDT_Double> ;
#endif



