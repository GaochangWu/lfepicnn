#include "hip/hip_runtime.h"
// @file im2row_gpu.cu
// @brief Stack image patches as matrix rows (GPU)
// @author Andrea Vedaldi

/*
Copyright (C) 2014-15 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "im2row6D.hpp"
#include "../datacu.hpp"
#include <iostream>

using namespace vl ;

/* ---------------------------------------------------------------- */
/*                                                           im2row */
/* ---------------------------------------------------------------- */

template <typename T> __global__ void
im2row_forward_kernel(T* stacked,
                      T const* data,
                       const int numPatchesX,
                       const int numPatchesY,
                       const int numPatchesXAn,
                       const int numPatchesYAn,
                       const int numPatchSlices,
                       const int width, const int height,
                       const int widthAn, const int heightAn,
                       const int windowWidth, const int windowHeight,
                       const int windowWidthAn, const int windowHeightAn,
                       const int strideX, const int strideY,
                       const int strideXAn, const int strideYAn,
                       const int padLeft, const int padTop,
                       const int padLeftAn, const int padTopAn)
{
  /* each kernel copies the pixels in an image patch for one channel */
  int index = threadIdx.x + blockIdx.x * blockDim.x ;
  if (index < numPatchSlices) {
    /*
     get the patch slice (x,y, xan, yan, z) to copy
     */
    int x = index ;
    int y = x / numPatchesX ;
    int xAn = y / numPatchesY ;
    int yAn = xAn / numPatchesXAn ;
    int z = yAn / numPatchesYAn ;
    x %= numPatchesX ;
    y %= numPatchesY ;
    xAn %= numPatchesXAn ;
    yAn %= numPatchesYAn ;

    /*
     pick the top-left corer of the patch slice in the input image
     */
    int x_data = x * strideX - padLeft ;
    int y_data = y * strideY - padTop ;
    int x_dataAn = xAn * strideXAn - padLeftAn ;
    int y_dataAn = yAn * strideYAn - padTopAn ;
    data += (((z * heightAn + y_dataAn) * widthAn + x_dataAn ) * height + y_data ) * width + x_data;

    /*
     pick the column of the stacked image which contains this patch,
     and move down along the column at the beginning of the patch slice
     */
    int patchSliceOffset = (windowWidth*windowHeight*windowWidthAn*windowHeightAn) * z ;
    stacked += (((numPatchesYAn * patchSliceOffset + yAn) * numPatchesXAn + xAn ) * numPatchesY + y)
                * numPatchesX + x;

    /*
     copy the patch slice
     */
    for (int s = 0 ; s < windowHeightAn ; s += 1) {
      for (int t = 0 ; t < windowWidthAn ; t += 1) {
        for (int v = 0 ; v < windowHeight ; v += 1) {
          for (int u = 0 ; u < windowWidth ; u += 1) {
            if (y_data + v >= 0 &&
                y_data + v < height &&
                x_data + u >= 0 &&
                x_data + u < width &&
                y_dataAn + s >= 0 &&
                y_dataAn + s < heightAn &&
                x_dataAn + t >= 0 &&
                x_dataAn + t < widthAn) {
              *stacked = data[s * width * height * widthAn + t * width * height + v * width + u] ;
            } else {
              *stacked = 0 ;
            }
            stacked += (numPatchesX*numPatchesY*numPatchesXAn*numPatchesYAn) ;
          }
        }
      }
    }
  }
}

/* ---------------------------------------------------------------- */
/*                                           im2row backward kernel */
/* ---------------------------------------------------------------- */

// The next two functions assume b > 0.
__forceinline__ __device__
int floordiv6D(int a, int b)
{
  int q = a/b ;
  if (a >= 0 || a == q*b) return q ;
  return q - 1 ;
}

__forceinline__ __device__
int ceildiv6D(int a, int b)
{
  int q = a/b ;
  if (a <= 0 || a == q*b) return q ;
  return q + 1 ;
}


int floordiv6D_cpu(int a, int b)
{
  int q = a/b ;
  if (a >= 0 || a == q*b) return q ;
  return q - 1 ;
}

int ceildiv6D_cpu(int a, int b)
{
  int q = a/b ;
  if (a <= 0 || a == q*b) return q ;
  return q + 1 ;
}


template <typename T> __global__ void
im2row_backward_kernel(T* data,
                        T const* stacked,
                       const int numPatchesX,
                       const int numPatchesY,
                       const int numPatchesXAn,
                       const int numPatchesYAn,
                       const int dataVolume,
                       const int width,
                       const int height,
                       const int widthAn,
                       const int heightAn,
                       const int depth,
                       const int windowWidth,
                       const int windowHeight,
                       const int windowWidthAn,
                       const int windowHeightAn,
                       const int strideX,
                       const int strideY,
                       const int strideXAn,
                       const int strideYAn,
                       const int padLeft,
                       const int padTop,
                       const int padLeftAn,
                       const int padTopAn,
                       const int gcdx, const int gcdy,
                       const int xbar, const int ybar,
                       const int ubar, const int vbar,
                       const int gcdxAn, const int gcdyAn,
                       const int xbarAn, const int ybarAn,
                       const int ubarAn, const int vbarAn)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < dataVolume)
  {
    T accumulator = 0 ;
    /*
     The goal of this kernel is to accumulate data[index]=data[x_data,y_data]
     all elements of the patch matrix that received copies of data[index] in the forward
     pass. To do this, we need to find which patches (x,y) that contain
     copies of this pixel and the relative offsets (u,v) within each such
     patch.

     First, we find which patches (x,y) contain copies of pixel (x_data,y_data)
     in the input tensor. The input tensor coordiante (x_data,y_data) of
     pixel  (u,v) in patch (x,y) are related by equations:

       x_data = x * strideX + u * dilateX - padLeft,
       y_data = y * strideY + v * dilateY - padTop.

     Now we find all values of (x,y) that can be generated by this equation.
     These gives us the patches (x,y) that must be summed. We have:

       strideX * x + dilateX * u = x_data + padLeft.

     where x and u are integers. This is a linear Diophantine equation.
     Rewrite it as:

       ax + bu = c, where

       a = strideX,
       b = dilateY,
       c = x_data + padLeft.

     This equation has a solution only if the greatest common divisor
     g = gcd(a,b) of a and b divides c as well. In this case,
     let (x0,u0) be a solution (i.e. a x0 + b u0 = c); all other solutions
     are in the form

       x_k = x0 + Dx * k,  Dx = b/g,
       u_k = u0 - Du * k,  Du = a/g.

     Next, we look for the values of k such that x_k and u_k are within
     bounds:

       1) 0 <= x_k <= Pw - 1
       2) 0 <= u_k <= Ww - 1

     Thus

       0) recall: gcd(a,b) must divide c
       1) ceil(- x0/Dx) <= k <= floor((Iw - 1 - x0)/Dx)
       2) ceil((u0 - Ww + 1)/Du) <= k <= floor(u0/Du)

     Thus we need to look for the k in the interval

       k_min = ceil(max(-x0/Dx, (u0 - Ww + 1)/Du)),
       k_max = floor(min((Pw - 1 - x0)/Dx,u0/Du).

     Toghether with (*) and the corresponding equations for y,
     this produces a list of patches (x_k,y_p) that contains
     pixel (x_data,y_data) (the list can be empty).

     Furthermore, x_data is mapped to a specific pixel in
     patch x_k whose coordiante is u_k, also given above.
     */

    int x_data = index ;
    int y_data = x_data / width ;
    int x_dataAn = y_data / height ;
    int y_dataAn = x_dataAn / widthAn ;
    int z = y_dataAn / heightAn ;
    x_data %= width ;
    y_data %= height ;
    x_dataAn %= widthAn ;
    y_dataAn %= heightAn ;

    int cx = x_data + padLeft ;
    int cy = y_data + padTop ;
    int cxAn = x_dataAn + padLeftAn ;
    int cyAn = y_dataAn + padTopAn ;
    int qx = cx / gcdx ;
    int qy = cy / gcdy ;
    int qxAn = cxAn / gcdxAn ;
    int qyAn = cyAn / gcdyAn ;

    if (cx != gcdx * qx || cy != gcdy * qy || cxAn != gcdxAn * qxAn || cyAn != gcdyAn * qyAn) { data[index] = 0 ; return ; }

    int x0 = xbar * qx ;
    int u0 = ubar * qx ;
    int y0 = ybar * qy ;
    int v0 = vbar * qy ;
    int x0An = xbarAn * qxAn ;
    int u0An = ubarAn * qxAn ;
    int y0An = ybarAn * qyAn ;
    int v0An = vbarAn * qyAn ;

    int Dx = 1 / gcdx ;
    int Du = strideX / gcdx ;
    int Dy = 1 / gcdy ;
    int Dv = strideY / gcdy ;
    int DxAn = 1 / gcdxAn ;
    int DuAn = strideXAn / gcdxAn ;
    int DyAn = 1 / gcdyAn ;
    int DvAn = strideYAn / gcdyAn ; 
    
    int kmin1 = ceildiv6D(-x0,Dx) ;
    int kmax1 = floordiv6D(numPatchesX - 1 - x0,Dx) ;
    int kmin2 = ceildiv6D(u0 - windowWidth + 1,Du) ;
    int kmax2 = floordiv6D(u0,Du) ;
    int kmin = max(kmin1,kmin2) ;
    int kmax = min(kmax1,kmax2) ;

    int qmin1 = ceildiv6D(-y0,Dy) ;
    int qmax1 = floordiv6D(numPatchesY - 1 - y0,Dy) ;
    int qmin2 = ceildiv6D(v0 - windowHeight + 1,Dv) ;
    int qmax2 = floordiv6D(v0,Dv) ;
    int qmin = max(qmin1,qmin2) ;
    int qmax = min(qmax1,qmax2) ;

    int kmin1An = ceildiv6D(-x0An,DxAn) ;
    int kmax1An = floordiv6D(numPatchesXAn - 1 - x0An,DxAn) ;
    int kmin2An = ceildiv6D(u0An - windowWidthAn + 1,DuAn) ;
    int kmax2An = floordiv6D(u0An,DuAn) ;
    int kminAn = max(kmin1An,kmin2An) ;
    int kmaxAn = min(kmax1An,kmax2An) ;

    int qmin1An = ceildiv6D(-y0An,DyAn) ;
    int qmax1An = floordiv6D(numPatchesYAn - 1 - y0An,DyAn) ;
    int qmin2An = ceildiv6D(v0An - windowHeightAn + 1,DvAn) ;
    int qmax2An = floordiv6D(v0An,DvAn) ;
    int qminAn = max(qmin1An,qmin2An) ;
    int qmaxAn = min(qmax1An,qmax2An) ;

    /*
     Now we have kmin <= k <= kmax, qmin <= q <= qmax and

     x_k = x0 + Dx * k,     u_k = u0 - Du * k,
     y_q = y0 + Dy * q,     v_q = v0 - Dv * q.

     Thus for each (k,q) in the allowable range, we visit
     patch (x_k,y_q) and pixel (u_k,v_q) within it.

     (x_k,y_q) tells us which row of the patch matix to look for, and
     (u_k,v_q) tells us which column. Linearizing all this:

     pm_row(k,q) = y_q * numPatchesX + x_k,
     pm_col(k,q) = ((z * windowHeight) + v_q) * windowWidth + u_k.

     This is further linearized into an index:

     pm_index(k,q) = (numPatchesX*numPatchesY) * pm_col(k,q) + pm_row(k,q)

     Substituting everything

     pm_row(k,q)
     = (y0 + Dy * q) * numPatchesX + x0 + Dx * k
     = (numPatchesX * Dy) * q + Dx * k + (y0 * numPatchesX + x0)
     = rqc * q + rkc * k + roc

     pm_col(k,q)
     = ((z * windowHeight) + v0 - Dv * q) * windowWidth + u0 - Du * k
     = - (windowWidth * Dv) * q - (Du) * k + (windowHeight * windowWidth * z + v0 * windowWidth + u0)
     = cqc * q + ckc * k + coc ;

     pm_index(k,q)
     = (numPatchesX*numPatchesY) * (cqc * q + ckc * k + coc) + rqc * q + rkc * k + roc
     = (numPatchesX*numPatchesY * cqc + rqc) * q + (numPatchesX*numPatchesY * ckc + rkc) * k + (numPatchesX*numPatchesY * coc + roc)
     = iqc * q + ikc * k + ioc

     */
    
    int rqcAn = DyAn * numPatchesXAn * numPatchesX * numPatchesY ;
    int rkcAn = DxAn * numPatchesX * numPatchesY ;
    int rqc = numPatchesX * Dy ;
    int rkc = Dx ;
    int roc = y0An * numPatchesXAn * numPatchesX * numPatchesY + x0An * numPatchesX * numPatchesY + numPatchesX * y0 + x0 ;
    
    int cqcAn =  - windowWidthAn * windowHeight * windowWidth * DvAn;
    int ckcAn = - windowHeight * windowWidth * DuAn;
    int cqc = - windowWidth * Dv ;
    int ckc = - Du ;
    int coc = (( windowWidthAn * (windowHeightAn * z + v0An) + u0An ) * windowHeight + v0) * windowWidth + u0;

    int np = numPatchesX * numPatchesY * numPatchesXAn * numPatchesYAn ;
    int iqcAn = np * cqcAn + rqcAn ;
    int ikcAn = np * ckcAn + rkcAn ;
    int iqc = np * cqc + rqc ;
    int ikc = np * ckc + rkc ;
    int ioc = np * coc + roc ;

    stacked += ioc ;
    for (int qAn = qminAn ; qAn <= qmaxAn ; ++ qAn) {
      for (int kAn = kminAn ; kAn <= kmaxAn ; ++ kAn) {
        for (int q = qmin ; q <= qmax ; ++ q) {
          for (int k = kmin ; k <= kmax ; ++ k) {
            accumulator += stacked[iqcAn * qAn + ikcAn * kAn + iqc * q + ikc * k] ;
          }
        }
      }
    }
    data[index] = accumulator;
  }
}

namespace vl { namespace impl {

  template<typename type>
  struct im2row6D<vl::VLDT_GPU, type>
  {

    /* ------------------------------------------------------------ */
    /*                                                      forward */
    /* ------------------------------------------------------------ */

    static vl::ErrorCode
    forward(Context & context,
            type* stacked,
            type const* data,
            size_t width,
            size_t height,
            size_t widthAn,
            size_t heightAn,
            size_t depth,
            size_t windowWidth,
            size_t windowHeight,
            size_t windowWidthAn,
            size_t windowHeightAn,
            size_t strideX,
            size_t strideY,
            size_t strideXAn,
            size_t strideYAn,
            size_t padLeft,
            size_t padRight,
            size_t padTop,
            size_t padBottom,
            size_t padLeftAn,
            size_t padRightAn,
            size_t padTopAn,
            size_t padBottomAn)
    {
      /* Each kernel instance copies a feature dimension of a patch */


      int numPatchesX = (width + (padLeft + padRight) - windowWidth)/strideX + 1 ;
      int numPatchesY = (height + (padTop + padBottom) - windowHeight)/strideY + 1 ;
      int numPatchesXAn = (widthAn + (padLeftAn + padRightAn) - windowWidthAn)/strideXAn + 1 ;
      int numPatchesYAn = (heightAn + (padTopAn + padBottomAn) - windowHeightAn)/strideYAn + 1 ;
      int numPatchSlices = numPatchesX * numPatchesY * numPatchesXAn * numPatchesYAn * depth ;

      im2row_forward_kernel<type>
      <<< divideAndRoundUp(numPatchSlices, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
      (stacked,
       data,
       numPatchesX,
       numPatchesY,
       numPatchesXAn,
       numPatchesYAn,
       numPatchSlices,
       width, height,
       widthAn, heightAn,
       windowWidth, windowHeight,
       windowWidthAn, windowHeightAn,
       strideX, strideY,
       strideXAn, strideYAn,
       padLeft, padTop,
       padLeftAn, padTopAn) ;

      return context.setError(context.getCudaHelper().catchCudaError(__func__)) ;
    }

    /* ------------------------------------------------------------ */
    /*                                                     backward */
    /* ------------------------------------------------------------ */

    static vl::ErrorCode
    backward(Context & context,
             type* data,
             type const* stacked,
            size_t width,
            size_t height,
            size_t widthAn,
            size_t heightAn,
            size_t depth,
            size_t windowWidth,
            size_t windowHeight,
            size_t windowWidthAn,
            size_t windowHeightAn,
            size_t strideX,
            size_t strideY,
            size_t strideXAn,
            size_t strideYAn,
            size_t padLeft,
            size_t padRight,
            size_t padTop,
            size_t padBottom,
            size_t padLeftAn,
            size_t padRightAn,
            size_t padTopAn,
            size_t padBottomAn)
    {
      /*
       Each kernel integrates all contributions to a particular element
       of data.
       */

      int numPatchesX = (width + (padLeft + padRight) - windowWidth)/strideX + 1 ;
      int numPatchesY = (height + (padTop + padBottom) - windowHeight)/strideY + 1 ;
      int numPatchesXAn = (widthAn + (padLeftAn + padRightAn) - windowWidthAn)/strideXAn + 1 ;
      int numPatchesYAn = (heightAn + (padTopAn + padBottomAn) - windowHeightAn)/strideYAn + 1 ;
      int dataVolume = width * height * widthAn * heightAn * depth ;

      int xbar ;
      int ubar ;
      int gcdx = vl::gcd(strideX, 1, xbar, ubar) ;

      int ybar ;
      int vbar ;
      int gcdy = vl::gcd(strideY, 1, ybar, vbar) ;

      int xbarAn ;
      int ubarAn ;
      int gcdxAn = vl::gcd(strideXAn, 1, xbarAn, ubarAn) ;

      int ybarAn ;
      int vbarAn ;
      int gcdyAn = vl::gcd(strideYAn, 1, ybarAn, vbarAn) ;
      
      im2row_backward_kernel<type>
      <<< divideAndRoundUp(dataVolume, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
      (data,
       stacked,
       numPatchesX,
       numPatchesY,
       numPatchesXAn,
       numPatchesYAn,
       dataVolume,
       width, height, widthAn, heightAn, depth,
       windowWidth, windowHeight,
       windowWidthAn, windowHeightAn,
       strideX, strideY,
       strideXAn, strideYAn,
       padLeft, padTop,
       padLeftAn, padTopAn,
       gcdx, gcdy, xbar, ybar, ubar, vbar,
       gcdxAn, gcdyAn, xbarAn, ybarAn, ubarAn, vbarAn) ;

      return context.setError(context.getCudaHelper().catchCudaError(__func__)) ;
    }

  } ;

} }

// Instantiations
template struct vl::impl::im2row6D<vl::VLDT_GPU, float> ;

#ifdef ENABLE_DOUBLE
template struct vl::impl::im2row6D<vl::VLDT_GPU, double> ;
#endif
